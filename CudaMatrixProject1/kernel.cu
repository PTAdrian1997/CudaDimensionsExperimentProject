#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>
#include <stdio.h>

#define CHECK(_t, _e) if (_e != hipSuccess) { fprintf(stderr, "%s failed: %s", _t, hipGetErrorString(_e)); goto Error; }
#define HERR(_t, _e) if (_e != hipSuccess) { fprintf(stderr, "%s failed: %s", _t, hipGetErrorString(_e)); }

__global__ void getBlockDimAndGridDim(int * blockDimX, int * blockDimY, int * gridDimX, int * gridDimY) {
	*blockDimX = blockDim.x;
	*blockDimY = blockDim.y;
	*gridDimX = gridDim.x;
	*gridDimY = gridDim.y;
}

__global__ void init_device_a(int * device_a, int * max_index)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int offset = row * gridDim.x * blockDim.x + col;
	if(offset < *max_index)
		device_a[offset] = offset;
}

/* A helper function to print a matrix a with height and width*/
void print_a(const int * a, int height, int width) {
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++) printf("%d ", a[row * width + col]);
		printf("\n");
	}
}

void cudaGenerateMatrixFunction() {

	int HEIGHT = 8;
	int WIDTH = 16;

	int * a = (int *)malloc(HEIGHT * WIDTH * sizeof(int));

	int * device_a;
	CHECK("hipMalloc device_a", hipMalloc((void **)&device_a, HEIGHT * WIDTH * sizeof(int)));

	dim3 blockDimCust(8, 2);
	dim3 grid(8);
	int max_index = HEIGHT * WIDTH;
	int *device_max_index;

	CHECK("hipMalloc device_max_index", hipMalloc((void **)&device_max_index, sizeof(int)));
	CHECK("hipMemcpy device_max_index max_index", hipMemcpy(device_max_index, &max_index, sizeof(int), hipMemcpyHostToDevice));

	init_device_a <<<grid, blockDimCust>>> (device_a, device_max_index);

	CHECK("hipMemcpy a device_a", hipMemcpy(a, device_a, HEIGHT * WIDTH * sizeof(int), hipMemcpyDeviceToHost));

	print_a(a, HEIGHT, WIDTH);

Error:
	HERR("hipFree device_a", hipFree(device_a));
	HERR("hipFree device_max_index", hipFree(device_max_index));

}

void getDimensions() {
	int HEIGHT = 8;
	int WIDTH = 16;

	dim3 blockDimCust(8, 2);
	dim3 grid(8);

	int * deviceBlockDimX;
	int * deviceBlockDimY;
	int * deviceGridDimX;
	int * deviceGridDimY;

	CHECK("hipMalloc deviceBlockDimX", hipMalloc((void **)&deviceBlockDimX, sizeof(int)));
	CHECK("hipMalloc deviceBlockDimY", hipMalloc((void **)&deviceBlockDimY, sizeof(int)));
	CHECK("hipMalloc deviceGridDimX", hipMalloc((void **)&deviceGridDimX, sizeof(int)));
	CHECK("hipMalloc deviceGridDimY", hipMalloc((void **)&deviceGridDimY, sizeof(int)));

	getBlockDimAndGridDim<<<grid, blockDimCust>>>(deviceBlockDimX, deviceBlockDimY, deviceGridDimX, deviceGridDimY);

	int blockDimX, blockDimY, gridDimX, gridDimY;

	CHECK("hipMemcpy blockDimX deviceBlockDimX", hipMemcpy(&blockDimX, deviceBlockDimX, sizeof(int), hipMemcpyDeviceToHost));
	CHECK("hipMemcpy blockDimY deviceBlockDimY", hipMemcpy(&blockDimY, deviceBlockDimY, sizeof(int), hipMemcpyDeviceToHost));
	CHECK("hipMemcpy ", hipMemcpy(&gridDimX, deviceGridDimX, sizeof(int), hipMemcpyDeviceToHost));
	CHECK("hipMemcpy blockDimX deviceBlockDimX", hipMemcpy(&gridDimY, deviceGridDimY, sizeof(int), hipMemcpyDeviceToHost));

	printf("blockDimX = %d, blockDimY = %d, gridDimX = %d, gridDimY = %d\n", blockDimX, blockDimY, gridDimX, gridDimY);

Error:
	HERR("hipFree deviceBlockDimX", hipFree(deviceBlockDimX));
	HERR("hipFree deviceBlockDimY", hipFree(deviceBlockDimY));
	HERR("hipFree deviceGridDimX", hipFree(deviceGridDimX));
	HERR("hipFree deviceGridDimY", hipFree(deviceGridDimY));

}

int main()
{

	getDimensions();
	cudaGenerateMatrixFunction();

	return 0;
}